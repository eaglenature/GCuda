#include "hip/hip_runtime.h"
#include <gcuda/gcuda.h>
#include <thrust/host_vector.h>

template <typename T>
__global__ void incrementKernel(T* data, int size)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < size)
    {
        T d = data[tid];
        data[tid] = d + T(1);
    }
}

template <typename T>
__global__ void incrementWithErrorKernel(T* data, int size, double error)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < size)
    {
        T d = data[tid];
        data[tid] = d + error;
    }
}

//-------------------------------------------------------------//
//                                                             //
//                 ASSERT_DEVICE_ARRAY_EQ                      //
//                                                             //
//-------------------------------------------------------------//
TEST(AssertDeviceArrayEqual, Int)
{
    typedef int T;
    const int numElements = 16 * 128;
    thrust::host_vector<T> a(numElements);
    thrust::host_vector<T> ref(numElements);
    for (int i = 0; i < numElements; ++i)
    {
        a[i] = i;
        ref[i] = i + 1;
    }
    T* b;
    hipMalloc((void**)&b, sizeof(T) * numElements);
    hipMemcpy(b, a.data(), sizeof(T) * numElements, hipMemcpyHostToDevice);
    int threads = 128;
    int blocks = (numElements + threads - 1)/threads;
    incrementKernel<T><<<blocks, threads>>>(b, numElements);
    //ref[4] = 12;
    ASSERT_DEVICE_ARRAY_EQ(ref, b, numElements);
    hipFree(b);
}

TEST(AssertDeviceArrayEqual, Float)
{
    typedef float T;
    const int numElements = 16 * 128;
    std::vector<T> a(numElements);
    std::vector<T> ref(numElements);
    for (int i = 0; i < numElements; ++i)
    {
        a[i] = i;
        ref[i] = i + 1;
    }
    T* b;
    hipMalloc((void**)&b, sizeof(T) * numElements);
    hipMemcpy(b, a.data(), sizeof(T) * numElements, hipMemcpyHostToDevice);
    int threads = 128;
    int blocks = (numElements + threads - 1)/threads;
    incrementKernel<T><<<blocks, threads>>>(b, numElements);
    ASSERT_DEVICE_ARRAY_EQ(ref.data(), b, numElements);
    hipFree(b);
}


//-------------------------------------------------------------//
//                                                             //
//                 ASSERT_DEVICE_ARRAY_NEAR                    //
//                                                             //
//-------------------------------------------------------------//
TEST(AssertDeviceArrayNear, Float)
{
    typedef float T;
    const int numElements = 16 * 128;
    const double error = 0.00001;
    thrust::host_vector<T> a(numElements);
    thrust::host_vector<T> ref(numElements);
    for (int i = 0; i < numElements; ++i)
    {
        a[i] = i;
        ref[i] = i + error;
    }
    T* b;
    hipMalloc((void**)&b, sizeof(T) * numElements);
    hipMemcpy(b, a.data(), sizeof(T) * numElements, hipMemcpyHostToDevice);
    int threads = 128;
    int blocks = (numElements + threads - 1)/threads;
    incrementWithErrorKernel<T><<<blocks, threads>>>(b, numElements, error);
    const double absError = error * 1.1;
    ASSERT_DEVICE_ARRAY_NEAR(ref, b, numElements, absError);
    hipFree(b);
}

TEST(AssertDeviceArrayNear, Double)
{
    typedef double T;
    const int numElements = 16 * 128;
    const double error = 0.0000001;
    std::vector<T> a(numElements);
    std::vector<T> ref(numElements);
    for (int i = 0; i < numElements; ++i)
    {
        a[i] = i;
        ref[i] = i + error;
    }
    T* b;
    hipMalloc((void**)&b, sizeof(T) * numElements);
    hipMemcpy(b, a.data(), sizeof(T) * numElements, hipMemcpyHostToDevice);
    int threads = 128;
    int blocks = (numElements + threads - 1)/threads;
    incrementWithErrorKernel<T><<<blocks, threads>>>(b, numElements, error);
    const double absError = error * 1.01;
    ASSERT_DEVICE_ARRAY_NEAR(ref.data(), b, numElements, absError);
    hipFree(b);
}


//-------------------------------------------------------------//
//                                                             //
//                 EXPECT_DEVICE_ARRAY_EQ                      //
//                                                             //
//-------------------------------------------------------------//
TEST(ExpectDeviceArrayEqual, Int)
{
    typedef int T;
    const int numElements = 16 * 128;
    thrust::host_vector<T> a(numElements);
    thrust::host_vector<T> ref(numElements);
    for (int i = 0; i < numElements; ++i)
    {
        a[i] = i;
        ref[i] = i + 1;
    }
    T* b;
    hipMalloc((void**)&b, sizeof(T) * numElements);
    hipMemcpy(b, a.data(), sizeof(T) * numElements, hipMemcpyHostToDevice);
    int threads = 128;
    int blocks = (numElements + threads - 1)/threads;
    incrementKernel<T><<<blocks, threads>>>(b, numElements);
    //ref[4] = 12;
    EXPECT_DEVICE_ARRAY_EQ(ref, b, numElements);
    hipFree(b);
}

TEST(ExpectDeviceArrayEqual, Float)
{
    typedef float T;
    const int numElements = 16 * 128;
    std::vector<T> a(numElements);
    std::vector<T> ref(numElements);
    for (int i = 0; i < numElements; ++i)
    {
        a[i] = i;
        ref[i] = i + 1;
    }
    T* b;
    hipMalloc((void**)&b, sizeof(T) * numElements);
    hipMemcpy(b, a.data(), sizeof(T) * numElements, hipMemcpyHostToDevice);
    int threads = 128;
    int blocks = (numElements + threads - 1)/threads;
    incrementKernel<T><<<blocks, threads>>>(b, numElements);
    EXPECT_DEVICE_ARRAY_EQ(ref.data(), b, numElements);
    hipFree(b);
}


//-------------------------------------------------------------//
//                                                             //
//                 EXPECT_DEVICE_ARRAY_NEAR                    //
//                                                             //
//-------------------------------------------------------------//
TEST(ExpectDeviceArrayNear, Float)
{
    typedef float T;
    const int numElements = 16 * 128;
    const double error = 0.00001;
    thrust::host_vector<T> a(numElements);
    thrust::host_vector<T> ref(numElements);
    for (int i = 0; i < numElements; ++i)
    {
        a[i] = i;
        ref[i] = i + error;
    }
    T* b;
    hipMalloc((void**)&b, sizeof(T) * numElements);
    hipMemcpy(b, a.data(), sizeof(T) * numElements, hipMemcpyHostToDevice);
    int threads = 128;
    int blocks = (numElements + threads - 1)/threads;
    incrementWithErrorKernel<T><<<blocks, threads>>>(b, numElements, error);
    const double absError = error * 1.1;
    EXPECT_DEVICE_ARRAY_NEAR(ref, b, numElements, absError);
    hipFree(b);
}

TEST(ExpectDeviceArrayNear, Double)
{
    typedef double T;
    const int numElements = 16 * 128;
    const double error = 0.0000001;
    std::vector<T> a(numElements);
    std::vector<T> ref(numElements);
    for (int i = 0; i < numElements; ++i)
    {
        a[i] = i;
        ref[i] = i + error;
    }
    T* b;
    hipMalloc((void**)&b, sizeof(T) * numElements);
    hipMemcpy(b, a.data(), sizeof(T) * numElements, hipMemcpyHostToDevice);
    int threads = 128;
    int blocks = (numElements + threads - 1)/threads;
    incrementWithErrorKernel<T><<<blocks, threads>>>(b, numElements, error);
    const double absError = error * 1.01;
    EXPECT_DEVICE_ARRAY_NEAR(ref.data(), b, numElements, absError);
    hipFree(b);
}



int main(int argc, char **argv)
{
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}
